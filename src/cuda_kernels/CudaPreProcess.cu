#include "hip/hip_runtime.h"
//
// Created by lijin on 2023/12/21.
//

#include "CudaPreProcess.h"

#define GPU_BLOCK_THREADS 512

#define KernelPositionBlock                                 \
    int position = (blockDim.x * blockIdx.x + threadIdx.x); \
    if (position >= (edge))                                 \
        return;

#define checkCudaKernel(...)                                            \
    __VA_ARGS__;                                                        \
    do                                                                  \
    {                                                                   \
        hipError_t cudaStatus = hipPeekAtLastError();                 \
        if (cudaStatus != hipSuccess)                                  \
        {                                                               \
            INFOE("launch failed: %s", hipGetErrorString(cudaStatus)); \
        }                                                               \
    } while (0);

#define Assert(op)                        \
    do                                    \
    {                                     \
        bool cond = !(!(op));             \
        if (!cond)                        \
        {                                 \
            INFOF("Assert failed, " #op); \
        }                                 \
    } while (false)

namespace CUDA
{


    dim3 grid_dims(int numJobs)
    {
        int numBlockThreads = numJobs < GPU_BLOCK_THREADS ? numJobs : GPU_BLOCK_THREADS;
        return dim3(((numJobs + numBlockThreads - 1) / (float)numBlockThreads));
    }

    dim3 block_dims(int numJobs)
    {
        return numJobs < GPU_BLOCK_THREADS ? numJobs : GPU_BLOCK_THREADS;
    }

    Norm Norm::mean_std(const float mean[3],
                        const float std[3],
                        float alpha,
                        ChannelType channel_type)
    {
        Norm out;
        out.type = NormType::MeanStd;
        out.alpha = alpha;
        out.channel_type = channel_type;
        memcpy(out.mean, mean, sizeof(out.mean));
        memcpy(out.std, std, sizeof(out.std));
        return out;
    }

    Norm Norm::alpha_beta(float alpha, float beta, ChannelType channel_type)
    {
        Norm out;
        out.type = NormType::AlphaBeta;
        out.alpha = alpha;
        out.beta = beta;
        out.channel_type = channel_type;
        return out;
    }

#define INTER_RESIZE_COEF_BITS 11
#define INTER_RESIZE_COEF_SCALE (1 << INTER_RESIZE_COEF_BITS)
#define CAST_BITS (INTER_RESIZE_COEF_BITS << 1)
    template <typename _T>
    static __inline__ __device__ _T limit(_T value, _T low, _T high)
    {
        return value < low ? low : (value > high ? high : value);
    }

    static __inline__ __device__ int resize_cast(int value)
    {
        return (value + (1 << (CAST_BITS - 1))) >> CAST_BITS;
    }

    __global__ void crop_cvtcolor_kernel(uint8_t *output,
                                         const uint8_t *input,
                                         int x,
                                         int y,
                                         int width, int height,
                                         int input_width,
                                         int input_height,
                                         int format)
        {
            int x_new = blockIdx.x * blockDim.x + threadIdx.x;
            int y_new = blockIdx.y * blockDim.y + threadIdx.y;

            if (x_new < width && y_new < height)
            {
                int index_new = (y_new * width + x_new) * 3;
                int x_old = x + x_new;
                int y_old = y + y_new;
                if (x_old >= 0 && x_old < input_width && y_old >= 0 && y_old < input_height)
                {
                    int total_pixels = input_width * input_height;

                    int index_old = (y_old * input_width + x_old);
                    switch (format)
                    {
                    case ColorFormat::RGBP:
                        output[index_new + 0] = input[index_old + 0 * total_pixels]; // R
                        output[index_new + 1] = input[index_old + 1 * total_pixels]; // G
                        output[index_new + 2] = input[index_old + 2 * total_pixels]; // B
                        break;
                    case ColorFormat::RGB:
                        output[index_new + 0] = input[index_old * 3 + 0]; // R
                        output[index_new + 1] = input[index_old * 3 + 1]; // G
                        output[index_new + 2] = input[index_old * 3 + 2]; // B
                        break;
                    case ColorFormat::BGR:
                        output[index_new + 0] = input[index_old * 3 + 2]; // R
                        output[index_new + 1] = input[index_old * 3 + 1]; // G
                        output[index_new + 2] = input[index_old * 3 + 0]; // B
                        break;
                    }
                }
                else
                {
                    output[index_new + 0] = 0; // or other padding value
                    output[index_new + 1] = 0; // or other padding value
                    output[index_new + 2] = 0; // or other padding value
                }
            }
        }

        __global__ void resize_bilinear_and_normalize_kernel(uint8_t * src,
                                                             int src_line_size,
                                                             int src_width,
                                                             int src_height,
                                                             float *dst,
                                                             int dst_width,
                                                             int dst_height,
                                                             float sx,
                                                             float sy,
                                                             Norm norm,
                                                             int edge)
        {
            int position = blockDim.x * blockIdx.x + threadIdx.x;
            if (position >= edge)
                return;

            int dx = position % dst_width;
            int dy = position / dst_width;
            float src_x = (dx + 0.5f) * sx - 0.5f;
            float src_y = (dy + 0.5f) * sy - 0.5f;
            float c0, c1, c2;

            int y_low = floorf(src_y);
            int x_low = floorf(src_x);
            int y_high = limit(y_low + 1, 0, src_height - 1);
            int x_high = limit(x_low + 1, 0, src_width - 1);
            y_low = limit(y_low, 0, src_height - 1);
            x_low = limit(x_low, 0, src_width - 1);

            int ly = rint((src_y - y_low) * INTER_RESIZE_COEF_SCALE);
            int lx = rint((src_x - x_low) * INTER_RESIZE_COEF_SCALE);
            int hy = INTER_RESIZE_COEF_SCALE - ly;
            int hx = INTER_RESIZE_COEF_SCALE - lx;
            int w1 = hy * hx, w2 = hy * lx, w3 = ly * hx, w4 = ly * lx;
            //    float   *pdst = dst + dy * dst_width + dx * 3;
            uint8_t *v1 = src + y_low * src_line_size + x_low * 3;
            uint8_t *v2 = src + y_low * src_line_size + x_high * 3;
            uint8_t *v3 = src + y_high * src_line_size + x_low * 3;
            uint8_t *v4 = src + y_high * src_line_size + x_high * 3;

            c0 = resize_cast(w1 * v1[0] + w2 * v2[0] + w3 * v3[0] + w4 * v4[0]);
            c1 = resize_cast(w1 * v1[1] + w2 * v2[1] + w3 * v3[1] + w4 * v4[1]);
            c2 = resize_cast(w1 * v1[2] + w2 * v2[2] + w3 * v3[2] + w4 * v4[2]);

            if (norm.channel_type == ChannelType::Invert)
            {
                float t = c2;
                c2 = c0;
                c0 = t;
            }

            if (norm.type == NormType::MeanStd)
            {
                c0 = (c0 * norm.alpha - norm.mean[0]) / norm.std[0];
                c1 = (c1 * norm.alpha - norm.mean[1]) / norm.std[1];
                c2 = (c2 * norm.alpha - norm.mean[2]) / norm.std[2];
            }
            else if (norm.type == NormType::AlphaBeta)
            {
                c0 = c0 * norm.alpha + norm.beta;
                c1 = c1 * norm.alpha + norm.beta;
                c2 = c2 * norm.alpha + norm.beta;
            }

            int area = dst_width * dst_height;
            float *pdst_c0 = dst + dy * dst_width + dx;
            float *pdst_c1 = pdst_c0 + area;
            float *pdst_c2 = pdst_c1 + area;
            *pdst_c0 = c0;
            *pdst_c1 = c1;
            *pdst_c2 = c2;
        }

        __global__ void warp_perspective_kernel(uint8_t * src,
                                                int src_line_size,
                                                int src_width,
                                                int src_height,
                                                float *dst,
                                                int dst_width,
                                                int dst_height,
                                                uint8_t const_value_st,
                                                float *warp_affine_matrix_3_3,
                                                Norm norm,
                                                int edge)
        {
            int position = blockDim.x * blockIdx.x + threadIdx.x;
            if (position >= edge)
                return;

            float m_x1 = warp_affine_matrix_3_3[0];
            float m_y1 = warp_affine_matrix_3_3[1];
            float m_z1 = warp_affine_matrix_3_3[2];

            float m_x2 = warp_affine_matrix_3_3[3];
            float m_y2 = warp_affine_matrix_3_3[4];
            float m_z2 = warp_affine_matrix_3_3[5];

            float m_x3 = warp_affine_matrix_3_3[6];
            float m_y3 = warp_affine_matrix_3_3[7];
            float m_z3 = warp_affine_matrix_3_3[8];

            int dx = position % dst_width;
            int dy = position / dst_width;

            // 原图位置
            float src_x = (m_x1 * dx + m_y1 * dy + m_z1) / (m_x3 * dx + m_y3 * dy + m_z3);
            float src_y = (m_x2 * dx + m_y2 * dy + m_z2) / (m_x3 * dx + m_y3 * dy + m_z3);
            float c0, c1, c2;

            if (src_x <= -1 || src_x >= src_width || src_y <= -1 || src_y >= src_height)
            {
                // out of range
                c0 = const_value_st;
                c1 = const_value_st;
                c2 = const_value_st;
            }
            else
            {
                int y_low = floorf(src_y);
                int x_low = floorf(src_x);
                int y_high = y_low + 1;
                int x_high = x_low + 1;

                uint8_t const_value[] = {const_value_st, const_value_st, const_value_st};
                float ly = src_y - y_low;
                float lx = src_x - x_low;
                float hy = 1 - ly;
                float hx = 1 - lx;
                float w1 = hy * hx, w2 = hy * lx, w3 = ly * hx, w4 = ly * lx;
                uint8_t *v1 = const_value;
                uint8_t *v2 = const_value;
                uint8_t *v3 = const_value;
                uint8_t *v4 = const_value;
                if (y_low >= 0)
                {
                    if (x_low >= 0)
                        v1 = src + y_low * src_line_size + x_low * 3;

                    if (x_high < src_width)
                        v2 = src + y_low * src_line_size + x_high * 3;
                }

                if (y_high < src_height)
                {
                    if (x_low >= 0)
                        v3 = src + y_high * src_line_size + x_low * 3;

                    if (x_high < src_width)
                        v4 = src + y_high * src_line_size + x_high * 3;
                }

                // same to opencv
                c0 = floorf(w1 * v1[0] + w2 * v2[0] + w3 * v3[0] + w4 * v4[0] + 0.5f);
                c1 = floorf(w1 * v1[1] + w2 * v2[1] + w3 * v3[1] + w4 * v4[1] + 0.5f);
                c2 = floorf(w1 * v1[2] + w2 * v2[2] + w3 * v3[2] + w4 * v4[2] + 0.5f);
            }

            if (norm.channel_type == ChannelType::Invert)
            {
                float t = c2;
                c2 = c0;
                c0 = t;
            }

            if (norm.type == NormType::MeanStd)
            {
                c0 = (c0 * norm.alpha - norm.mean[0]) / norm.std[0];
                c1 = (c1 * norm.alpha - norm.mean[1]) / norm.std[1];
                c2 = (c2 * norm.alpha - norm.mean[2]) / norm.std[2];
            }
            else if (norm.type == NormType::AlphaBeta)
            {
                c0 = c0 * norm.alpha + norm.beta;
                c1 = c1 * norm.alpha + norm.beta;
                c2 = c2 * norm.alpha + norm.beta;
            }

            int area = dst_width * dst_height;
            float *pdst_c0 = dst + dy * dst_width + dx;
            float *pdst_c1 = pdst_c0 + area;
            float *pdst_c2 = pdst_c1 + area;
            *pdst_c0 = c0;
            *pdst_c1 = c1;
            *pdst_c2 = c2;
        }

        __global__ void warp_affine_bilinear_and_normalize_plane_kernel(uint8_t * src,
                                                                        int src_line_size,
                                                                        int src_width,
                                                                        int src_height,
                                                                        float *dst,
                                                                        int dst_width,
                                                                        int dst_height,
                                                                        uint8_t const_value_st,
                                                                        float *warp_affine_matrix_2_3,
                                                                        Norm norm,
                                                                        int edge)
        {
            int position = blockDim.x * blockIdx.x + threadIdx.x;
            if (position >= edge)
                return;

            float m_x1 = warp_affine_matrix_2_3[0];
            float m_y1 = warp_affine_matrix_2_3[1];
            float m_z1 = warp_affine_matrix_2_3[2];
            float m_x2 = warp_affine_matrix_2_3[3];
            float m_y2 = warp_affine_matrix_2_3[4];
            float m_z2 = warp_affine_matrix_2_3[5];

            int dx = position % dst_width;
            int dy = position / dst_width;
            float src_x = m_x1 * dx + m_y1 * dy + m_z1;
            float src_y = m_x2 * dx + m_y2 * dy + m_z2;
            float c0, c1, c2;

            if (src_x <= -1 || src_x >= src_width || src_y <= -1 || src_y >= src_height)
            {
                // out of range
                c0 = const_value_st;
                c1 = const_value_st;
                c2 = const_value_st;
            }
            else
            {
                int y_low = floorf(src_y);
                int x_low = floorf(src_x);
                int y_high = y_low + 1;
                int x_high = x_low + 1;

                uint8_t const_value[] = {const_value_st, const_value_st, const_value_st};
                float ly = src_y - y_low;
                float lx = src_x - x_low;
                float hy = 1 - ly;
                float hx = 1 - lx;
                float w1 = hy * hx, w2 = hy * lx, w3 = ly * hx, w4 = ly * lx;
                uint8_t *v1 = const_value;
                uint8_t *v2 = const_value;
                uint8_t *v3 = const_value;
                uint8_t *v4 = const_value;
                if (y_low >= 0)
                {
                    if (x_low >= 0)
                        v1 = src + y_low * src_line_size + x_low * 3;

                    if (x_high < src_width)
                        v2 = src + y_low * src_line_size + x_high * 3;
                }

                if (y_high < src_height)
                {
                    if (x_low >= 0)
                        v3 = src + y_high * src_line_size + x_low * 3;

                    if (x_high < src_width)
                        v4 = src + y_high * src_line_size + x_high * 3;
                }

                // same to opencv
                c0 = floorf(w1 * v1[0] + w2 * v2[0] + w3 * v3[0] + w4 * v4[0] + 0.5f);
                c1 = floorf(w1 * v1[1] + w2 * v2[1] + w3 * v3[1] + w4 * v4[1] + 0.5f);
                c2 = floorf(w1 * v1[2] + w2 * v2[2] + w3 * v3[2] + w4 * v4[2] + 0.5f);
            }

            if (norm.channel_type == ChannelType::Invert)
            {
                float t = c2;
                c2 = c0;
                c0 = t;
            }

            if (norm.type == NormType::MeanStd)
            {
                c0 = (c0 - norm.mean[0]) / norm.std[0] * norm.alpha;
                c1 = (c1 - norm.mean[1]) / norm.std[1] * norm.alpha;
                c2 = (c2 - norm.mean[2]) / norm.std[2] * norm.alpha;
            }
            else if (norm.type == NormType::AlphaBeta)
            {
                c0 = c0 * norm.alpha + norm.beta;
                c1 = c1 * norm.alpha + norm.beta;
                c2 = c2 * norm.alpha + norm.beta;
            }

            int area = dst_width * dst_height;
            float *pdst_c0 = dst + dy * dst_width + dx;
            float *pdst_c1 = pdst_c0 + area;
            float *pdst_c2 = pdst_c1 + area;
            *pdst_c0 = c0;
            *pdst_c1 = c1;
            *pdst_c2 = c2;
        }

        __global__ void warp_affine_bilinear_and_normalize_focus_kernel(uint8_t * src,
                                                                        int src_line_size,
                                                                        int src_width,
                                                                        int src_height,
                                                                        float *dst,
                                                                        int dst_width,
                                                                        int dst_height,
                                                                        uint8_t const_value_st,
                                                                        float *warp_affine_matrix_1_3,
                                                                        Norm norm,
                                                                        int edge)
        {
            int position = blockDim.x * blockIdx.x + threadIdx.x;
            if (position >= edge)
                return;

            float m_k = *warp_affine_matrix_1_3++;
            float m_b0 = *warp_affine_matrix_1_3++;
            float m_b1 = *warp_affine_matrix_1_3++;

            int dx = position % dst_width;
            int dy = position / dst_width;
            float src_x = m_k * dx + m_b0;
            float src_y = m_k * dy + m_b1;
            float c0, c1, c2;

            if (src_x <= -1 || src_x >= src_width || src_y <= -1 || src_y >= src_height)
            {
                // out of range
                c0 = const_value_st;
                c1 = const_value_st;
                c2 = const_value_st;
            }
            else
            {
                int y_low = floorf(src_y);
                int x_low = floorf(src_x);
                int y_high = y_low + 1;
                int x_high = x_low + 1;

                uint8_t const_value[] = {const_value_st, const_value_st, const_value_st};
                float ly = src_y - y_low;
                float lx = src_x - x_low;
                float hy = 1 - ly;
                float hx = 1 - lx;
                float w1 = hy * hx, w2 = hy * lx, w3 = ly * hx, w4 = ly * lx;
                uint8_t *v1 = const_value;
                uint8_t *v2 = const_value;
                uint8_t *v3 = const_value;
                uint8_t *v4 = const_value;
                if (y_low >= 0)
                {
                    if (x_low >= 0)
                        v1 = src + y_low * src_line_size + x_low * 3;

                    if (x_high < src_width)
                        v2 = src + y_low * src_line_size + x_high * 3;
                }

                if (y_high < src_height)
                {
                    if (x_low >= 0)
                        v3 = src + y_high * src_line_size + x_low * 3;

                    if (x_high < src_width)
                        v4 = src + y_high * src_line_size + x_high * 3;
                }

                // same to opencv
                c0 = floorf(w1 * v1[0] + w2 * v2[0] + w3 * v3[0] + w4 * v4[0] + 0.5f);
                c1 = floorf(w1 * v1[1] + w2 * v2[1] + w3 * v3[1] + w4 * v4[1] + 0.5f);
                c2 = floorf(w1 * v1[2] + w2 * v2[2] + w3 * v3[2] + w4 * v4[2] + 0.5f);
            }

            if (norm.channel_type == ChannelType::Invert)
            {
                float t = c2;
                c2 = c0;
                c0 = t;
            }

            if (norm.type == NormType::MeanStd)
            {
                c0 = (c0 * norm.alpha - norm.mean[0]) / norm.std[0];
                c1 = (c1 * norm.alpha - norm.mean[1]) / norm.std[1];
                c2 = (c2 * norm.alpha - norm.mean[2]) / norm.std[2];
            }
            else if (norm.type == NormType::AlphaBeta)
            {
                c0 = c0 * norm.alpha + norm.beta;
                c1 = c1 * norm.alpha + norm.beta;
                c2 = c2 * norm.alpha + norm.beta;
            }

            int after_focus_width = dst_width / 2;
            int after_focus_height = dst_height / 2;
            int fdx = dx / 2;
            int fdy = dy / 2;
            int fc = ((dx % 2) << 1) | (dy % 2);

            /**
             *   x[..., ::2, ::2], x[..., 1::2, ::2], x[..., ::2, 1::2], x[..., 1::2, 1::2]
             *    4                     fc
             *    3                     [0, 1, 2]
             *    after_focus_height    fdy
             *    after_focus_width     fdx
             *    左乘右加
             **/

            float *pdst_c0 = dst + ((fc * 3 + 0) * after_focus_height + fdy) * after_focus_width + fdx;
            float *pdst_c1 = dst + ((fc * 3 + 1) * after_focus_height + fdy) * after_focus_width + fdx;
            float *pdst_c2 = dst + ((fc * 3 + 2) * after_focus_height + fdy) * after_focus_width + fdx;

            *pdst_c0 = c0;
            *pdst_c1 = c1;
            *pdst_c2 = c2;
        }

        __global__ void bgr_to_gray_kernel(uint8_t * src, int src_line_size, float *dst, int edge)
        {
            int position = blockDim.x * blockIdx.x + threadIdx.x;
            if (position >= edge)
                return;

            uint8_t *psrc = src + position * 3;
            float c0 = psrc[0];
            float c1 = psrc[1];
            float c2 = psrc[2];
            dst[position] = 0.299f * c2 + 0.587f * c1 + 0.114f * c0;
        }

        __global__ void
        normalize_feature_kernel(float *feature_array, int num_feature, int feature_length, int edge)
        {
            /*
            &   1 gz         bi.z   0
            *   1 gy         bi.y   0
            *   N NF         bi.x   ~
            *   1 1          ti.z   0
            *   F FL / 32    ti.y   ~
            *   Q 32         ti.x   ~
            */

            int position = (blockIdx.x * blockDim.y + threadIdx.y) * blockDim.x + threadIdx.x;
            if (position >= edge)
                return;

            extern __shared__ float l2_norm[];

            int irow = position / feature_length;
            int icol = position % feature_length;

            if (icol == 0)
                l2_norm[irow] = 0;

            __syncthreads();

            float value = feature_array[position];
            atomicAdd(l2_norm + irow, value * value);

            __syncthreads();
            if (icol == 0)
                l2_norm[irow] = sqrt(l2_norm[irow]);

            __syncthreads();
            feature_array[position] = value / l2_norm[irow];
        }

        static __global__ void convert_nv12_to_bgr_kernel(const uint8_t *y,
                                                          const uint8_t *uv,
                                                          int width,
                                                          int height,
                                                          int linesize,
                                                          uint8_t *dst_bgr,
                                                          int edge)
        {
            int position = blockDim.x * blockIdx.x + threadIdx.x;
            if (position >= edge)
                return;

            int ox = position % width;
            int oy = position / width;
            const uint8_t &yvalue = y[oy * linesize + ox];
            int offset_uv = (oy >> 1) * linesize + (ox & 0xFFFFFFFE);
            const uint8_t &u = uv[offset_uv + 0];
            const uint8_t &v = uv[offset_uv + 1];
            dst_bgr[position * 3 + 0] = 1.164f * (yvalue - 16.0f) + 2.018f * (u - 128.0f);
            dst_bgr[position * 3 + 1] =
                1.164f * (yvalue - 16.0f) - 0.813f * (v - 128.0f) - 0.391f * (u - 128.0f);
            dst_bgr[position * 3 + 2] = 1.164f * (yvalue - 16.0f) + 1.596f * (v - 128.0f);
        }

        static __global__ void permute_NTCHW_to_NCTHW(float* output, const float* input, int N, int T, int C, int H, int W)
        {
            int index = blockIdx.x * blockDim.x + threadIdx.x;
            int total_size = N * T * C * H * W;

            if (index < total_size)
            {
                int n = index / (T * C * H * W);
                int idx2 = index % (T * C * H * W);
                int t = idx2 / (C * H * W);
                idx2 %= (C * H * W);
                int c = idx2 / (H * W);
                idx2 %= (H * W);
                int h = idx2 / W;
                int w = idx2 % W;

                int output_index = n * (C * T * H * W) + c * (T * H * W) + t * (H * W) + h * W + w;              
                
                output[output_index] = input[index];
            }
        }

        void permute_CT(float* output, const float* input, int N, int C, int T, int H, int W)
        {
            int total_size = N * C * T * H * W;
            int threads_per_block = 256;
            int num_blocks = (total_size + threads_per_block - 1) / threads_per_block;

            permute_NTCHW_to_NCTHW<<<num_blocks, threads_per_block>>>(output, input, N, C, T, H, W);
        }
        /////////////////////////////////////////////////////////////////////////
        /////////////////////////////////////////////////////////////////////////
        /////////////////////////////////////////////////////////////////////////
        /////////////////////////////////////////////////////////////////////////

        void crop_cvtcolor_Invoker(uint8_t * src,
                                   int src_line_size,
                                   int src_width,
                                   int src_height,
                                   uint8_t *dst,
                                   int x,
                                   int y,
                                   int width,
                                   int height,
                                   int format,
                                   hipStream_t stream)
        {
            dim3 block_size(16, 16);
            dim3 grid_size((width + block_size.x - 1) / block_size.x, (height + block_size.y - 1) / block_size.y);
            crop_cvtcolor_kernel<<<grid_size, block_size, 0, stream>>>(dst, src, x, y, width, height, src_width, src_height, format);
        }

        void convertNV12ToBgrInvoker(const uint8_t *y,
                                     const uint8_t *uv,
                                     int width,
                                     int height,
                                     int linesize,
                                     uint8_t *dst,
                                     hipStream_t stream)
        {
            int total = width * height;
            dim3 grid = grid_dims(total);
            dim3 block = block_dims(total);

            convert_nv12_to_bgr_kernel<<<grid, block, 0, stream>>>(y, uv, width, height, linesize, dst,
                                                                   total);
        }

        void warpAffineBilinearAndNormalizePlaneInvoker(uint8_t * src,
                                                        int src_line_size,
                                                        int src_width,
                                                        int src_height,
                                                        float *dst,
                                                        int dst_width,
                                                        int dst_height,
                                                        float *matrix_2_3,
                                                        uint8_t const_value,
                                                        const Norm &norm,
                                                        hipStream_t stream)
        {
            int jobs = dst_width * dst_height;
            auto grid = grid_dims(jobs);
            auto block = block_dims(jobs);

            warp_affine_bilinear_and_normalize_plane_kernel<<<grid, block, 0, stream>>>(
                src, src_line_size, src_width, src_height, dst, dst_width, dst_height, const_value,
                matrix_2_3, norm, jobs);
        }

        void warpAffineBilinearAndNormalizeFocusInvoker(uint8_t * src,
                                                        int src_line_size,
                                                        int src_width,
                                                        int src_height,
                                                        float *dst,
                                                        int dst_width,
                                                        int dst_height,
                                                        float *matrix_1_3,
                                                        uint8_t const_value,
                                                        const Norm &norm,
                                                        hipStream_t stream)
        {
            int jobs = dst_width * dst_height;
            auto grid = grid_dims(jobs);
            auto block = block_dims(jobs);

            warp_affine_bilinear_and_normalize_focus_kernel<<<grid, block, 0, stream>>>(
                src, src_line_size, src_width, src_height, dst, dst_width, dst_height, const_value,
                matrix_1_3, norm, jobs);
        }

        void warpPerspectiveInvoker(uint8_t * src,
                                    int src_line_size,
                                    int src_width,
                                    int src_height,
                                    float *dst,
                                    int dst_width,
                                    int dst_height,
                                    float *matrix_3_3,
                                    uint8_t const_value,
                                    const Norm &norm,
                                    hipStream_t stream)
        {
            int jobs = dst_width * dst_height;
            auto grid = grid_dims(jobs);
            auto block = block_dims(jobs);

            warp_perspective_kernel<<<grid, block, 0, stream>>>(src, src_line_size, src_width, src_height,
                                                                dst, dst_width, dst_height, const_value,
                                                                matrix_3_3, norm, jobs);
        }

        void resizeBilinearAndNormalizeInvoker(uint8_t * src,
                                               int src_line_size,
                                               int src_width,
                                               int src_height,
                                               float *dst,
                                               int dst_width,
                                               int dst_height,
                                               const Norm &norm,
                                               hipStream_t stream)
        {
            int jobs = dst_width * dst_height;
            auto grid = grid_dims(jobs);
            auto block = block_dims(jobs);

            resize_bilinear_and_normalize_kernel<<<grid, block, 0, stream>>>(
                src, src_line_size, src_width, src_height, dst, dst_width, dst_height,
                src_width / (float)dst_width, src_height / (float)dst_height, norm, jobs);
        }

        void normFeatureInvoker(float *feature_array,
                                int num_feature,
                                int feature_length,
                                hipStream_t stream)
        {
            if (feature_length % 32 != 0)
            {
                std::cout << "feature_length % 32 != 0" << std::endl;
                return;
            }

            int jobs = num_feature * feature_length;
            auto grid = dim3(num_feature);
            auto block = dim3(feature_length / 32, 32);
            normalize_feature_kernel<<<grid, block, num_feature * sizeof(float), stream>>>(
                feature_array, num_feature, feature_length, jobs);
        }

        void bgr2grayInvoker(uint8_t * src, float *dst, int width, int height, hipStream_t stream)
        {
            int jobs = width * height;
            auto grid = grid_dims(jobs);
            auto block = block_dims(jobs);

            bgr_to_gray_kernel<<<grid, block, 0, stream>>>(src, width * 3, dst, jobs);
        }

    } // namespace CUDA